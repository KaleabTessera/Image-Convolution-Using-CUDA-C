#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "helpers.h"


const char *imageFilename = "image21.pgm";

void imageConvolution(int argc, char **argv);
void printImage(float *image,int width,int height);
void printKernel(float * kernel, int kernelDimension);
void applyKernelToImageSerial(float * image, int imageWidth, int imageHeight, float * kernel, int kernelDimension, char *imagePath);
void flipKernel(float* kernel, int kernelDimension);
void loadKernels(float * kernel, char buf[512]);
void loadAllKernels(float ** kernels,  FILE* fp);
int getNumKernels(FILE* fp);
<<<<<<< HEAD
float applyKernelPerPixel(int y, int x,int kernelX, int kernelY, int imageWidth, int imageHeight, float *kernel, float *image);
void applyKernelToImageParallelNaive(float * image, int imageWidth, int imageHeight, float * kernel, int kernelDimension, char *imagePath);
__global__ void applyKernelPerPixelParallel(int * kernelX, int * kernelY, int * imageWidth, int * imageHeight, float * kernel, float * image,float * sumArray);
=======
float applyKernelPerPixel(int y, int x,int kernelX, int kernelY, int imageWidth, int imageHeight, float * kernel,float *image);
void applyKernelToImageParallelNaive(float * image, int imageWidth, int imageHeight, float * kernel, int kernelDimension, char *imagePath);
>>>>>>> 7351ef51280844a84d3c1bdc98103d7670d541d2

int main(int argc, char **argv)
{
  imageConvolution(argc,argv);
  return 0;
}


void imageConvolution(int argc, char **argv)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
	// load image from disk
    float *hData = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename,argv[0]);

    if (imagePath == NULL)
    {
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(imagePath, &hData, &width, &height);
    printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);
    
    //Get Kernels
    FILE* fp = fopen("kernels.txt", "r");
    if(fp == NULL) {
      perror("Error in opening file");
      exit(EXIT_FAILURE);
   }

    int numKernels = getNumKernels(fp);
    //printf("%d",numKernels);
    int kernelDimension = 3;
    
    float **kernels= (float**)malloc(sizeof(float*)*numKernels);
    for(int i =0; i < numKernels;i++ ){
      kernels[i] =  (float*)malloc(sizeof(float)*100);
    }
    loadAllKernels(kernels,fp);
    fclose(fp);
    hipEventRecord(start);
    //Flip kernels to match convolution property and apply kernels to image
    for(int i =0; i < numKernels;i++ ){
<<<<<<< HEAD
      applyKernelToImageParallelNaive(hData, width, height,kernels[i],kernelDimension,imagePath);
=======
      applyKernelToImageSerial(hData, width, height,kernels[i],kernelDimension,imagePath);
>>>>>>> 7351ef51280844a84d3c1bdc98103d7670d541d2
    } 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f \n",milliseconds);
}

void applyKernelToImageParallelNaive(float * image, int imageWidth, int imageHeight, float * kernel, int kernelDimension, char *imagePath){
  int *d_kernelDimensionX,*d_kernelDimensionY,*d_imageWidth,*d_imageHeight;
  float *d_kernel,*d_image,*d_sumArray;
  
  float *sumArray = (float*)malloc(imageWidth*imageHeight);
  int sizeInt = sizeof(int);
  int sizeFloat = sizeof(float);

  hipMalloc((void **)&d_kernelDimensionX,sizeInt);
  hipMalloc((void **)&d_kernelDimensionY,sizeInt);
  hipMalloc((void **)&d_imageWidth,sizeInt);
  hipMalloc((void **)&d_imageHeight,sizeInt);
  hipMalloc((void **)&d_kernel,sizeFloat);
  hipMalloc((void **)&d_image,sizeFloat);
  hipMalloc((void **)&d_sumArray,sizeFloat);

  hipMemcpy(d_kernelDimensionX,&kernelDimension,sizeInt,hipMemcpyHostToDevice);
  hipMemcpy(d_kernelDimensionY,&kernelDimension,sizeInt,hipMemcpyHostToDevice);
  hipMemcpy(d_imageWidth,&imageWidth,sizeInt,hipMemcpyHostToDevice);
  hipMemcpy(d_imageHeight,&imageHeight,sizeInt,hipMemcpyHostToDevice);
  hipMemcpy(d_kernel,&kernel,sizeFloat,hipMemcpyHostToDevice);
  hipMemcpy(d_image,&image,sizeFloat,hipMemcpyHostToDevice);

  dim3 gridNumber( imageHeight,imageWidth );
  dim3 threadsPerBlock( 512);
  //printf("%f \n", kernel[0]); 
  applyKernelPerPixelParallel<<<8,160>>>(d_kernelDimensionX,d_kernelDimensionY,d_imageWidth,d_imageHeight, d_kernel,d_image,d_sumArray);

  hipMemcpy(&sumArray,d_sumArray,sizeFloat,hipMemcpyHostToDevice);
  printf("%f \n",sumArray[0]);
  unsigned int size = imageWidth * imageHeight * sizeof(float);
  float *newImage = (float *) malloc(size);

  printImage(newImage,imageWidth,imageHeight,"newImage.txt");
  char outputFilename[1024];
  strcpy(outputFilename, imagePath);
  strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
  sdkSavePGM(outputFilename, newImage, imageWidth, imageHeight);
}

void applyKernelToImageSerial(float * image, int imageWidth, int imageHeight, float * kernel, int kernelDimension, char *imagePath){
  unsigned int size = imageWidth * imageHeight * sizeof(float);
  float *newImage = (float *) malloc(size);
  for(int y =0; y < imageHeight; y++){
    for(int x=0; x < imageWidth; x++){
      float sum = applyKernelPerPixel(y,x,kernelDimension,kernelDimension,imageWidth,imageHeight, kernel,image);
      //Normalising output - image doesn't get brighter or dimmer
       newImage[y*imageWidth+x] = sum/(kernelDimension * kernelDimension);
    }
  }
  printImage(newImage,imageWidth,imageHeight,"newImage.txt");
  char outputFilename[1024];
  strcpy(outputFilename, imagePath);
  strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
  sdkSavePGM(outputFilename, newImage, imageWidth, imageHeight);
}

float applyKernelPerPixel(int y, int x,int kernelX, int kernelY, int imageWidth, int imageHeight, float *kernel, float *image){
  float sum = 0;
  int offsetX = (kernelX - 1) / 2;
  int offsetY = (kernelY - 1) / 2;

  for (int j = 0; j < kernelY; j++) {
    //Ignore out of bounds
    if (y + j < offsetY
            || y + j - offsetY >= imageHeight)
            continue;

       for (int i = 0; i < kernelX; i++) {
         //Ignore out of bounds
         if (x + i < offsetX
                    || x + i - offsetX >= imageWidth)
            continue;

         float k = kernel[i + j * kernelY];
         float imageElement =  image[y*imageWidth+x + i - offsetX + imageWidth*(j-1)];
         float value = k * imageElement;
         sum = sum +value; 
       }     
      }
      return sum;
}

__global__ void applyKernelPerPixelParallel(int * d_kernelDimensionX, int * d_kernelDimensionY, int * d_imageWidth, int * d_imageHeight, float * d_kernel, float * d_image,float * d_sumArray){
  int x= threadIdx.x;
  int y= threadIdx.y;
  //printf("%d \n",x); 
  //printf("%d \n",y); 
  int offsetX = (*d_kernelDimensionX - 1) / 2;
  int offsetY = (*d_kernelDimensionY - 1) / 2;
  float sumy =0;
  for (int j = 0; j < *d_kernelDimensionY; j++) {
    //Ignore out of bounds
    if (y + j < offsetY
            || y + j - offsetY >= *d_imageHeight)
            continue;

       for (int i = 0; i < *d_kernelDimensionX; i++) {
         //Ignore out of bounds
         if (x + i < offsetX
                    || x + i - offsetX >= *d_imageWidth)
            continue;

         float k = d_kernel[i + j * (*d_kernelDimensionY)];
        //  printf("%d \n",(i + j * (*d_kernelDimensionY))); 
         printf("%f \n", d_kernel[0]); 
         float imageElement =  d_image[y* (*d_imageWidth)+x + i - offsetX + (*d_imageWidth)*(j-1)];
         float value = k * imageElement;
         sumy = sumy +value; 
        
       } 
       //printf("%f \n",sumy);    
      }
      
      d_sumArray[y+x] = sumy;
      //return sum;
}
