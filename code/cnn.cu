#include <iostream>
#include <stdio.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "helpers.h"
#include "imageConvolutionSerial.h"
#include "imageConvolutionParallel.h"


const char *imageFilename = "image21.pgm";

int main(int argc, char **argv)
{
  printf("Image convolution project \n");
  printf("Please select an option \n");
  printf("1 - Serial Implementation \n");
  printf("2 - Naive parallel implementation \n");
  
  int option;
  scanf("%d", &option);

  switch(option) {
    case 1  :
      imageConvolutionSerial(imageFilename,argv);
      break; 
   
    case 2  :
       imageConvolutionParallel(imageFilename,argv);
       break; 
   
    default : 
      printf("Incorrect input \n");
 }

  return 0;
}


