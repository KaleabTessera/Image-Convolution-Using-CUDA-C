#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "helpers.h"


const char *imageFilename = "image21.pgm";

void imageConvolution(int argc, char **argv);
void printImage(float *image,int width,int height);
void printKernel(float * kernel, int kernelDimension);
void applyKernelToImageSerial(float * image, int imageWidth, int imageHeight, float * kernel, int kernelDimension, char *imagePath);
void flipKernel(float* kernel, int kernelDimension);
void loadKernels(float * kernel, char buf[512]);
void loadAllKernels(float ** kernels,  FILE* fp);
int getNumKernels(FILE* fp);
float applyKernelPerPixel(int y, int x,int kernelX, int kernelY, int imageWidth, int imageHeight, float * kernel,float *image);
void applyKernelToImageParallelNaive(float * image, int imageWidth, int imageHeight, float * kernel, int kernelDimension, char *imagePath);

int main(int argc, char **argv)
{
  imageConvolution(argc,argv);
  return 0;
}


void imageConvolution(int argc, char **argv)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
	// load image from disk
    float *hData = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename,argv[0]);

    if (imagePath == NULL)
    {
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(imagePath, &hData, &width, &height);
    printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);
    
    //Get Kernels
    FILE* fp = fopen("kernels.txt", "r");
    if(fp == NULL) {
      perror("Error in opening file");
      exit(EXIT_FAILURE);
   }

    int numKernels = getNumKernels(fp);
    //printf("%d",numKernels);
    int kernelDimension = 3;
    
    float **kernels= (float**)malloc(sizeof(float*)*numKernels);
    for(int i =0; i < numKernels;i++ ){
      kernels[i] =  (float*)malloc(sizeof(float)*100);
    }
    loadAllKernels(kernels,fp);
    fclose(fp);
    hipEventRecord(start);
    //Flip kernels to match convolution property and apply kernels to image
    for(int i =0; i < numKernels;i++ ){
      applyKernelToImageSerial(hData, width, height,kernels[i],kernelDimension,imagePath);
    } 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f \n",milliseconds);
}

void applyKernelToImageParallelNaive(float * image, int imageWidth, int imageHeight, float * kernel, int kernelDimension, char *imagePath){
  unsigned int size = imageWidth * imageHeight * sizeof(float);
  float *newImage = (float *) malloc(size);
  for(int y =0; y < imageHeight; y++){
    for(int x=0; x < imageWidth; x++){

      float sum = applyKernelPerPixel(y,x,kernelDimension,kernelDimension,imageWidth,imageHeight, kernel,image);
      //Normalising output - image doesn't get brighter or dimmer
       newImage[y*imageWidth+x] = sum/(kernelDimension * kernelDimension);
    }
  }
  printImage(newImage,imageWidth,imageHeight,"newImage.txt");
  char outputFilename[1024];
  strcpy(outputFilename, imagePath);
  strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
  sdkSavePGM(outputFilename, newImage, imageWidth, imageHeight);
}

void applyKernelToImageSerial(float * image, int imageWidth, int imageHeight, float * kernel, int kernelDimension, char *imagePath){
  unsigned int size = imageWidth * imageHeight * sizeof(float);
  float *newImage = (float *) malloc(size);
  for(int y =0; y < imageHeight; y++){
    for(int x=0; x < imageWidth; x++){
      float sum = applyKernelPerPixel(y,x,kernelDimension,kernelDimension,imageWidth,imageHeight, kernel,image);
      //Normalising output - image doesn't get brighter or dimmer
       newImage[y*imageWidth+x] = sum/(kernelDimension * kernelDimension);
    }
  }
  printImage(newImage,imageWidth,imageHeight,"newImage.txt");
  char outputFilename[1024];
  strcpy(outputFilename, imagePath);
  strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
  sdkSavePGM(outputFilename, newImage, imageWidth, imageHeight);
}

float applyKernelPerPixel(int y, int x,int kernelX, int kernelY, int imageWidth, int imageHeight, float *kernel, float *image){
  float sum = 0;
  int offsetX = (kernelX - 1) / 2;
  int offsetY = (kernelY - 1) / 2;

  for (int j = 0; j < kernelY; j++) {
    //Ignore out of bounds
    if (y + j < offsetY
            || y + j - offsetY >= imageHeight)
            continue;

       for (int i = 0; i < kernelX; i++) {
         //Ignore out of bounds
         if (x + i < offsetX
                    || x + i - offsetX >= imageWidth)
            continue;

         float k = kernel[i + j * kernelY];
         float imageElement =  image[y*imageWidth+x + i - offsetX + imageWidth*(j-1)];
         float value = k * imageElement;
         sum = sum +value; 
       }     
      }
      return sum;
}

__global__ void applyKernelPerPixelParallel(int y, int x,int kernelX, int kernelY, int imageWidth, int imageHeight, float *kernel, float *image){
  float sum = 0;
  int offsetX = (kernelX - 1) / 2;
  int offsetY = (kernelY - 1) / 2;

  for (int j = 0; j < kernelY; j++) {
    //Ignore out of bounds
    if (y + j < offsetY
            || y + j - offsetY >= imageHeight)
            continue;

       for (int i = 0; i < kernelX; i++) {
         //Ignore out of bounds
         if (x + i < offsetX
                    || x + i - offsetX >= imageWidth)
            continue;

         float k = kernel[i + j * kernelY];
         float imageElement =  image[y*imageWidth+x + i - offsetX + imageWidth*(j-1)];
         float value = k * imageElement;
         sum = sum +value; 
       }     
      }
      //return sum;
}
